#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include "matrix_utils.h"

__global__ void kernel(int* matrix, int* res_matrix, const int size) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cnt = 0;
	if (tid < size) {
		//count positive
		for (int i = 0; i < size; i++)
			if (matrix[tid * size + i] > 0)
				cnt += 1;

		//write shifted
		for (int i = 0; i < size; i++) {
			res_matrix[tid * size + ((i + cnt) % size)] = matrix[tid * size + i];
		}
	}
}



int main(int argc, char* argv[]) {

	FILE* in_file;

	int* matrix = NULL;
	int* res_matrix = NULL;
	int* g_matrix = NULL;
	int* g_res_matrix = NULL;
	int size;

	in_file = fopen(argv[1], "r");
	if (in_file == NULL)
	{
		puts("Open in file error");
		return -1;
	}

	fscanf(in_file, "%d", &size);

	matrix = (int*)malloc(size * size * sizeof(int));
	res_matrix = (int*)malloc(size * size * sizeof(int));

	matrix = read_flat_matrix(in_file, size, matrix);

	hipMalloc((void**)&g_matrix, sizeof(int) * size * size);
	hipMalloc((void**)&g_res_matrix, sizeof(int) * size * size);

	printf("CUDA allocated memory for result vector, matrix and input vector\n");

	hipMemcpy(g_matrix, matrix, sizeof(int) * size * size, hipMemcpyHostToDevice);
	hipMemcpy(g_res_matrix, res_matrix, sizeof(int) * size * size, hipMemcpyHostToDevice);
	printf("CUDA copied input matrix and vector\n");

	print_flat_matrix(size, matrix);

	printf("Running kernel with N = %d\n", size);


	kernel << <size, 1 >> > (g_matrix, g_res_matrix, size);
	

	hipMemcpy(res_matrix, g_res_matrix, sizeof(int) * size * size, hipMemcpyDeviceToHost);
	print_flat_matrix(size, res_matrix);

	hipFree(g_matrix);
	hipFree(g_res_matrix);


	FILE* out_file;

	out_file = fopen(argv[2], "w");
	if (out_file == NULL)
	{
		puts("Open out file error");
		return -1;
	}

	write_flat_matrix(out_file, res_matrix, size, size);

	free(matrix);
	free(res_matrix);

	fclose(out_file);

	return 0;
};
