#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include "matrix_utils.h"

__global__ void kernel(int* matrix, int* res_matrix, const int m, const int size) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cnt = 0;
	if (tid < m) {
		//count positive
		for (int i = 0; i < size; i++)
			if (matrix[tid * size + i] > 0)
				cnt += 1;

		//write shifted
		for (int i = 0; i < size; i++) {
			res_matrix[tid * size + ((i + cnt) % size)] = matrix[tid * size + i];
		}
	}
}



int* shift_matrix_cuda(int* matrix, int m, int size) {

	int* res_matrix = NULL;
	int* g_matrix = NULL;
	int* g_res_matrix = NULL;

	res_matrix = (int*)malloc(m * size * sizeof(int));


	hipMalloc((void**)&g_matrix, sizeof(int) * m * size);
	hipMalloc((void**)&g_res_matrix, sizeof(int) * m * size);


	printf("CUDA allocated memory for result vector, matrix and input vector\n");

	hipMemcpy(g_matrix, matrix, sizeof(int) * m * size, hipMemcpyHostToDevice);
	hipMemcpy(g_res_matrix, res_matrix, sizeof(int) * m * size, hipMemcpyHostToDevice);
	printf("CUDA copied input matrix and vector\n");


	printf("Running kernel with N = %d\n", size);


	kernel << <size, 1 >> > (g_matrix, g_res_matrix, m, size);
	
	hipMemcpy(res_matrix, g_res_matrix, sizeof(int) * m * size, hipMemcpyDeviceToHost);
	print_flat_matrix(size, res_matrix);

	hipFree(g_matrix);
	hipFree(g_res_matrix);

	return res_matrix;
};
