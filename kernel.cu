#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include <fstream>

__global__ void kernel(int* matrix, int* res_matrix, const int m, const int size) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cnt = 0;
	if (tid < m) {
		//count positive
		for (int i = 0; i < size; i++)
			if (matrix[tid * size + i] > 0)
				cnt += 1;

		//write shifted
		for (int i = 0; i < size; i++) {
			res_matrix[tid * size + ((i + cnt) % size)] = matrix[tid * size + i];
		}
	}
}



int* shift_matrix_cuda(int* matrix, int m, int size) {

	int* res_matrix = NULL;
	int* g_matrix = NULL;
	int* g_res_matrix = NULL;

	res_matrix = (int*)malloc(m * size * sizeof(int));


	hipMalloc((void**)&g_matrix, sizeof(int) * m * size);
	hipMalloc((void**)&g_res_matrix, sizeof(int) * m * size);


	printf("CUDA allocated memory for result vector, matrix block\n");

	hipMemcpy(g_matrix, matrix, sizeof(int) * m * size, hipMemcpyHostToDevice);
	hipMemcpy(g_res_matrix, res_matrix, sizeof(int) * m * size, hipMemcpyHostToDevice);
	printf("CUDA copied input matrix block\n");


	printf("Running kernel with N = %d\n", size);


	kernel << <256, 1 >> > (g_matrix, g_res_matrix, m, size);
	
	hipMemcpy(res_matrix, g_res_matrix, sizeof(int) * m * size, hipMemcpyDeviceToHost);

	hipFree(g_matrix);
	hipFree(g_res_matrix);

	return res_matrix;
};
